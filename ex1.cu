/* C stuff */
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include <string.h>
#include <errno.h>

// C++ stuff
#include <iostream>
#include <fstream>
#include <string>
#include <iomanip>
#include <sstream>

// Open-CV for the vision stuff
//#include <opencv2/opencv.hpp>

/* Cuda stuff */
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

typedef unsigned char byte;
typedef byte * pbyte;

clock_t LastProfilingClock=clock();

#define ARCH_NEWLINE	"\n"

/***************************************************************************
 Writes profiling output (milli-seconds since last call)
 ***************************************************************************/

extern clock_t LastProfilingClock;

inline float profiling (const char *s, clock_t *whichClock=NULL) 
{
	if (whichClock==NULL)
		whichClock=&LastProfilingClock;

    clock_t newClock=clock();
    float res = (float) (newClock-*whichClock) / (float) CLOCKS_PER_SEC;
    if (s!=NULL)
        std::cerr << "Time: " << s << ": " << res << std::endl; 
    *whichClock = newClock;
    return res;
}

inline float profilingTime (const char *s, time_t *whichClock) 
{
    time_t newTime=time(NULL);
    float res = (float) (newTime-*whichClock);
    if (s!=NULL)
        std::cerr << "Time(real): " << s << ": " << res << std::endl; 
    return res;
}

/***************************************************************************
 CREATES AN EMPTY IMAGE
 ***************************************************************************/

unsigned char **CREATE_IMAGE (int ysize, int xsize)	{
	unsigned char ** im;
	unsigned char *big;

	im = new pbyte [xsize];
	big	= new byte [xsize*ysize];

	for	(int i = 0 ; i < xsize ; i++)
		im[i] =	big	+ i*ysize;	

	return (im);
}

/***************************************************************************
 Frees an image
 ***************************************************************************/

void FREE_IMAGE	(byte **im)	
{
	delete [] im[0];
	delete [] im;
}

/***************************************************************************
 Reads a grayscale image
 ***************************************************************************/

void readImage (const char *filename, unsigned char***_p2darr, int *_ysize, int *_xsize) {

	char *buf;
	char shortbuf[256];
	short int x, y;
	int	color, foo;
	char c;
	FILE * inpic;
	int	entete,	z;
	int ysize, xsize;
	unsigned char **R;

	if ( (inpic	= fopen(filename,"r+b")) == NULL)	
	{
		std::cerr << "can't open file '" << filename << "': " << strerror(errno) << std::endl;
		exit(1);
	}

	if (fscanf(inpic,"%c%c\n",&c,&c) !=	2) 
	{
		std::cerr <<  "Image::readGray():\n Wrong Image Format: no .ppm!!\n"
  			 << "filename: " << filename << std::endl;
  		exit(2);
	}

	if (c == '6')  
	{
		z =	3 ;
		std::cerr << "Image::readGray():: disabled due to bug.\n"
			"Use Image::readColor() + Image::convertRGB2GrayScale() instead\n";
		exit(3);
	}
	else
	{
		if (c != '5') {
			std::cerr <<  "Image::readGray():: wrong image format: "
				"for .ppm only versions P5 and P6 are supported!\n";
			exit(4);
		}
		z =	1 ;
	}

	fscanf(inpic,"%c",&c) ;
	entete = 3 ;
	while (c ==	'#') {
		entete++ ;
		while (c !=	'\n') {
			entete++ ;
			fscanf(inpic,"%c",&c) ;
		}
		fscanf(inpic,"%c",&c) ;
	}

	if ( (inpic	= freopen(filename,"r+b",inpic)) == NULL)	{
		std::cerr << "can't open file " << filename << ":" << strerror(errno) << "\n";
		exit(5);
	}
	fread(shortbuf,1,entete,inpic);

	if (fscanf(inpic,"%d%d\n%d",&xsize,&ysize,&color) != 3)	{
		std::cerr << "Internal error (2):" << filename << std::endl;
		exit(6);
	}

	fread(shortbuf,1,1,inpic) ;

	buf	= new char [z*xsize+10];

	R =	CREATE_IMAGE(ysize,xsize) ;
	for	( y	= 0	; y	< ysize	; y++) 
	{

		if ((foo=fread(buf,1,z*xsize,inpic)) != z*xsize) 
		{
			std::ostringstream s;
			s << "file " << filename << ":\nrow " << y << " input failure: "
				<< "got " << foo << " instead of " << z*xsize << " bytes!\n";
			
			if (!feof(inpic))
				s << "No ";			
			s << "EOF occured.\n";
			if (!ferror(inpic))
				s << "No ";			
			std::cerr << "error in the sense of ferror() occured.\n";
			exit(7);
		}
		else 
		{
			if (z == 1)	
			{
				for	( x	= 0	; x	< xsize	; x++)
					R[x][y]	= buf[x] ;
			}
			else 
			{
				for	( x	= 0	; x	< z*xsize ;	x += z )
					R[x/z][y] =	(int)(.299*(float)buf[x] + 0.587*(float)buf[x+1]
						+ 0.114*(float)buf[x+2]);
			}
		}
	}
	fclose (inpic);
	delete [] buf;

	*_ysize = ysize;
	*_xsize = xsize;
	*_p2darr = R;
}

// *************************************************************
// Writes a	grayscale image
// *************************************************************

void writeImage(const char *filename, unsigned char **R, int ysize, int xsize) 
{
	FILE *fp;
	char *buf;
	short int y, x;

    if ((fp=fopen(filename,"w+b"))==NULL) 
    {
    	std::cerr << "Cannot create output file '" << filename << "': " << strerror(errno) << "!\n";
    	exit(1);
    }
    		
	buf = new char [xsize+10];

	sprintf(buf,"P5%s%d	%d%s255%s",ARCH_NEWLINE,xsize,ysize,ARCH_NEWLINE,ARCH_NEWLINE)	;
	x =	strlen(buf);
	clearerr(fp);
	fwrite(buf,1,x,fp);
	if (ferror(fp)) 
	{
		std::cerr << "Could not write image to file (Image::writeGray())!\n";
		exit(1);
	}

	for	( y	= 0	; y	< ysize	; y++)	{
		for	( x	= 0	; x	< xsize	; x++ )	{
			buf[x] = R[x][y];
		}

		clearerr(fp);
		fwrite(buf,1,xsize,fp);
		if (ferror(fp))
		{
			std::cerr << "Could not write image to file (Image::writeGray())!\n";
			exit(1);
		}
	}
	delete [] buf;
	fclose(fp);
}

/***************************************************************************
 USAGE
 ***************************************************************************/

void usage (char *com) 
{
    std::cerr<< "usage: " << com << " <inputimagename> <outputimagename>\n";
    exit(1);
}

/***************************************************************************
 The CPU version
 ***************************************************************************/

void cpuFilter(unsigned char *in, unsigned char *out, int rows, int cols)
{

	// General case
	for (int y=1; y<rows-1; ++y)
	for (int x=1; x<cols-1; ++x)
	{
		float f = (
			4.0*in[x*rows+y] +
			2.0*in[(x-1)*rows+y] +
			2.0*in[(x+2)*rows+y] +
			2.0*in[x*rows+y+1] +
			2.0*in[x*rows+y-1] +
			in[(x-1)*rows+y-1] +
			in[(x-1)*rows+y+1] +
			in[(x+1)*rows+y-1] +
			in[(x+1)*rows+y+1]
			)/16.0;
		if (f<0) f=0;
		if (f>255) f=255;
		out[x*rows+y] = (unsigned char) f;
	}
	
	// Borders
	for (int y=0; y<rows; ++y)
	{
		out[0*rows+y] = in[0*rows+y];
		out[(cols-1)*rows+y] = in[(cols-1)*rows+y];
	}
		
	for (int x=0; x<cols; ++x)
	{
		out[x*rows+0] = in[x*rows+0];
		out[x*rows+rows-1] = in[x*rows+rows-1];
	}
}

/***************************************************************************
 The GPU version - the kernel
 ***************************************************************************/

__global__
void gpuHostRun(int mxWidth, unsigned char* input, unsigned char* output)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x; // cols
	int y = blockIdx.y*blockDim.y + threadIdx.y; // rows
	if(y*mxWidth + x <= mxWidth*mxWidth)
	{
		if(!(y == mxWidth-1 || y == 0 || x == mxWidth-1 || x == 0)){	
		
			float f = (
				4.0*input[x*mxWidth+y] +
				2.0*input[(x-1)*mxWidth+y] +
				2.0*input[(x+2)*mxWidth+y] +
				2.0*input[x*mxWidth+y+1] +
				2.0*input[x*mxWidth+y-1] +
				input[(x-1)*mxWidth+y-1] +
				input[(x-1)*mxWidth+y+1] +
				input[(x+1)*mxWidth+y-1] +
				input[(x+1)*mxWidth+y+1]
				)/16.0;
			if (f<0) f=0;
			if (f>255) f=255;
			output[x*mxWidth+y] = (unsigned char) f;
		}
		else {
			output[x*mxWidth+y] = input[x*mxWidth+y];		
		}
	}
}


 /***************************************************************************
 The GPU version - the host code
 ***************************************************************************/

void gpuFilter(unsigned char *imarr, unsigned char *resarr, int rows, int cols ) // dimY == nbRows, dimX == nbCol
{
	unsigned char *gpuMatrix1; //input
	unsigned char *gpuMatrix2; //output
	

	int matrixInByte = rows*cols*sizeof(char);
	
	hipMalloc((void**) &gpuMatrix1, matrixInByte);
	hipMalloc((void**) &gpuMatrix2, matrixInByte);

	hipError_t ok = hipMemcpy(gpuMatrix1, imarr, matrixInByte, hipMemcpyHostToDevice );
	if(ok != hipSuccess)
	{
		std::cerr <<"*** Could not transfer\n";
		exit(1);
	}

	dim3 dimBlock(32,32);
	dim3 dimGrid(cols/dimBlock.x,rows/dimBlock.y);

	gpuHostRun<<<dimGrid, dimBlock>>>(cols, gpuMatrix1, gpuMatrix2);

	hipMemcpy(resarr, gpuMatrix2, matrixInByte, hipMemcpyDeviceToHost );
	if(ok != hipSuccess)
	{
		std::cerr <<"*** Could not transfer\n";
		exit(1);
	}
	
}

	
/***************************************************************************
 Main program
 ***************************************************************************/


int main (int argc, char **argv)
{
	int c;
	// Argument processing
    while ((c =	getopt (argc, argv,	"h")) != EOF) 
    {
		switch (c) {

			case 'h':
				usage(*argv);
				break;
	
			case '?':
				usage (*argv);
				std::cerr << "\n" << "*** Problem parsing the options!\n\n";
				exit (1);
		}
	}	

    int requiredArgs=2;

	if (argc-optind!=requiredArgs) 
    {
        usage (*argv);
		exit (1);
	}
	char *inputfname=argv[optind];
	char *outputfname=argv[optind+1];

	std::cout << "Reading image " << inputfname << "\n";

	unsigned char **image;
	int rows; 
	int cols;
	readImage (inputfname, &image, &rows, &cols);
	
	std::cout << "=====================================================\n"
		<< "Loaded image of size " << cols << "x" << rows << ".\n";

	unsigned char *imarr = *image;
	unsigned char *resarr = new unsigned char [cols*rows];

	profiling (NULL);
	
	for (int i=0; i<100; ++i)
		cpuFilter(imarr, resarr, rows, cols);

	profiling ("CPU version");

	for (int i=0; i<100; ++i)
		gpuFilter(imarr, resarr, rows, cols);
	
	profiling ("GPU version");

	// Copy flat array back to image structure
	for (int i=0; i<rows*cols; ++i)
		imarr[i] = resarr[i];
	
	writeImage (outputfname, image, rows, cols);
	
    std::cout << "Program terminated correctly.\n";
    return 0;
}

